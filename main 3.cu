/*
 * Title: CS6023, GPU Programming, Jan-May 2023, Assignment-1
 * Description: Computation of a matrix C = Kronecker_prod(A, B.T)
 *              where A and B are matrices of dimension (m, n) and
 *              the output is of the dimension (m * n, m * n).
 * Note: All lines marked in --> should be replaced with code.
 */

#include <cstdio>     // Added for printf() function
#include <sys/time.h> // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
using namespace std;

ofstream outfile; // The handle for printing the output

__global__ void per_row_AB_kernel(long int *A, long int *B, long int *C, long int m, long int n)
{
    // --> Complete the kernel ....
    long int p = blockIdx.x;
    long int q = threadIdx.x;
    for (long int i = 0; i < n; i++)
    {
        for (long int j = 0; j < n; j++)
        {
            C[(j + p * n) * m * n + q + i * m] = A[p * n + i] * B[q * n + j];
        }
    }
}

__global__ void per_column_AB_kernel(long int *A, long int *B, long int *C, long int m, long int n)
{
    // --> Complete the kernel ....
    long int p = blockIdx.x;
    long int q = threadIdx.x * blockDim.y + threadIdx.y;
    if (q < n)
    {
        for (long int i = 0; i < m; i++)
        {
            for (long int j = 0; j < m; j++)
            {
                C[(p + i * n) * m * n + m * q + j] = A[i * n + q] * B[j * n + p];
            }
        }
    }
}

__global__ void per_element_kernel(long int *A, long int *B, long int *C, long int m, long int n)
{
    // --> Complete the kernel ....
    long int id = (blockDim.x * blockDim.y) * (blockIdx.x * gridDim.y + blockIdx.y) + (threadIdx.x * blockDim.y + threadIdx.y);
    if (id < (m * m * n * n))
    {
        long int i = id / (m * n);
        long int j = id % (int)(m * n);
        C[i * m * n + j] = A[(i / n) * n + (j / m)] * B[(j % (int)m) * n + (i % (int)n)];
    }
}

/**
 * Prints any 1D array in the form of a matrix
 **/
void printMatrix(long int *arr, long int rows, long int cols, char *filename)
{
    outfile.open(filename);
    for (long int i = 0; i < rows; i++)
    {
        for (long int j = 0; j < cols; j++)
        {
            outfile << arr[i * cols + j] << " ";
        }
        outfile << "\n";
    }
    outfile.close();
}

/**
 * Timing functions taken from the matrix multiplication source code
 * rtclock - Returns the time of the day
 * printtime - Prints the time taken for computation
 **/
double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0)
        printf("Error return from gettimeofday: %d", stat);
    return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime)
{
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main(int argc, char **argv)
{
    // Variable declarations
    long int m, n;
    cin >> m >> n;

    // Host_arrays
    long int *h_a, *h_b, *h_c;

    // Device arrays
    long int *d_a, *d_b, *d_c;

    // Allocating space for the host_arrays
    h_a = (long int *)malloc(m * n * sizeof(long int));
    h_b = (long int *)malloc(m * n * sizeof(long int));
    h_c = (long int *)malloc(m * m * n * n * sizeof(long int));

    // Allocating memory for the device arrays
    // --> Allocate memory for A on device
    hipMalloc(&d_a, m * n * sizeof(long int));

    // --> Allocate memory for B on device
    hipMalloc(&d_b, m * n * sizeof(long int));

    // --> Allocate memory for C on device
    hipMalloc(&d_c, m * m * n * n * sizeof(long int));

    // Read the input matrix A
    for (long int i = 0; i < m * n; i++)
    {
        cin >> h_a[i];
    }

    // Read the input matrix B
    for (long int i = 0; i < m * n; i++)
    {
        cin >> h_b[i];
    }

    // Transfer the input host arrays to the device
    // --> Copy A from Host to Device
    hipMemcpy(d_a, h_a, m * n * sizeof(long int), hipMemcpyHostToDevice);
    // --> Copy B from Host to Device
    hipMemcpy(d_b, h_b, m * n * sizeof(long int), hipMemcpyHostToDevice);

    long int gridDimx, gridDimy;

    // Launch the kernels
    /**
     * Kernel 1 - per_row_AB_kernel
     * To be launched with 1D grid, 1D block
     * Each thread should process a complete row of A, B
     **/

    // --> Set the launch configuration
    dim3 grid1(m, 1, 1);
    dim3 block1(m, 1, 1);

    double starttime = rtclock();

    // --> Launch the kernel
    per_row_AB_kernel<<<grid1, block1>>>(d_a, d_b, d_c, m, n);

    hipDeviceSynchronize();

    double endtime = rtclock();
    printtime("GPU Kernel-1 time: ", starttime, endtime);

    // --> Copy C from Device to Host
    hipMemcpy(h_c, d_c, m * m * n * n * sizeof(long int), hipMemcpyDeviceToHost);

    printMatrix(h_c, m * n, m * n, "kernel1.txt");
    hipMemset(d_c, 0, m * n * m * n * sizeof(long int));

    /**
     * Kernel 2 - per_column_AB_kernel
     * To be launched with 1D grid, 2D block
     * Each thread should process a complete column of  A, B
     **/

    // --> Set the launch configuration
    gridDimy = ceil(float(n) / 5);
    dim3 grid2(n, 1, 1);
    dim3 block2(5, gridDimy, 1);

    starttime = rtclock();

    // --> Launch the kernel
    per_column_AB_kernel<<<grid2, block2>>>(d_a, d_b, d_c, m, n);

    hipDeviceSynchronize();

    endtime = rtclock();
    printtime("GPU Kernel-2 time: ", starttime, endtime);

    // --> Copy C from Device to Host
    hipMemcpy(h_c, d_c, m * m * n * n * sizeof(long int), hipMemcpyDeviceToHost);

    printMatrix(h_c, m * n, m * n, "kernel2.txt");
    hipMemset(d_c, 0, m * n * m * n * sizeof(long int));

    /**
     * Kernel 3 - per_element_kernel
     * To be launched with 2D grid, 2D block
     * Each thread should process one element of the output
     **/
    gridDimx = ceil(float(n * n) / 16);
    gridDimy = ceil(float(m * m) / 64);
    dim3 grid3(gridDimx, gridDimy, 1);
    dim3 block3(64, 16, 1);

    starttime = rtclock();

    // --> Launch the kernel
    per_element_kernel<<<grid3, block3>>>(d_a, d_b, d_c, m, n);

    hipDeviceSynchronize();

    endtime = rtclock();
    printtime("GPU Kernel-3 time: ", starttime, endtime);

    // --> Copy C from Device to Host
    hipMemcpy(h_c, d_c, m * m * n * n * sizeof(long int), hipMemcpyDeviceToHost);

    printMatrix(h_c, m * n, m * n, "kernel3.txt");

    return 0;
}